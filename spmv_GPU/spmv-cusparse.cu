#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "common.h"

const char* version_name = "cuSPARSE SpMV";\

#define CHECK_CUSPARSE(ret) if(ret != HIPSPARSE_STATUS_SUCCESS) { fprintf(stderr, "error in line %d\n", __LINE__);}

typedef struct {
    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void *dBuffer;
} additional_info_t;

typedef additional_info_t *info_ptr_t;

void preprocess(dist_matrix_t *mat, data_t *x, data_t *y) {
    info_ptr_t p = (info_ptr_t)malloc(sizeof(additional_info_t));
    hipsparseCreate(&p->handle);
    hipsparseCreateCsr(&p->matA, mat->global_m, mat->global_m, mat->global_nnz, mat->gpu_r_pos, mat->gpu_c_idx, mat->gpu_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&p->vecX, mat->global_m, x, HIP_R_32F);
    hipsparseCreateDnVec(&p->vecY, mat->global_m, y, HIP_R_32F);
    size_t buffersize;
    data_t alpha, beta;
    alpha = 1.0;
    beta = 1.0;
    hipsparseSpMV_bufferSize(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, p->matA, p->vecX, &beta, p->vecY, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, &buffersize);
    p->dBuffer = NULL;
    hipMalloc(&p->dBuffer, buffersize);
    //hipsparseSetMatIndexBase(p->descrA, HIPSPARSE_INDEX_BASE_ZERO);
    //hipsparseSetMatType(p->descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    mat->additional_info = p;
}

void destroy_additional_info(void *additional_info) {
    info_ptr_t p = (info_ptr_t)additional_info;
    hipsparseDestroySpMat(p->matA);
    hipsparseDestroyDnVec(p->vecX);
    hipsparseDestroyDnVec(p->vecY);
    hipsparseDestroy(p->handle);
    hipFree(p->dBuffer);
    free(p);
}

void spmv(dist_matrix_t *mat, const data_t* x, data_t* y) {
    int m = mat->global_m, nnz = mat->global_nnz;
    const data_t alpha = 1.0, beta = 1.0;
    info_ptr_t p = (info_ptr_t)mat->additional_info;

    hipsparseSpMV(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, p->matA, p->vecX, &beta, p->vecY, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, p->dBuffer);
}
